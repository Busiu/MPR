#include "hip/hip_runtime.h"
// Matrix multiplication by parts
// Elements stored in row-major order

using namespace std;
#include <stdio.h>
#include <iostream>
#include <fstream>
#include "helper_timer.h"
#define BLOCK_SIZE 16

typedef struct
{	int width;
	int height;
	float *elements;
} Matrix;

// Forward declaration of matrix mult
__global__ void MatMulKernel (const Matrix, const Matrix, Matrix);

// Host code
void MatMulGPU(const Matrix A, const Matrix B, Matrix C)
{
	StopWatchInterface *timer;

	// Load matrices A and B to device memory
	Matrix d_A;
	d_A.width = A.width; d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc((void**) &d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	
	Matrix d_B;
	d_B.width = B.width; d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void**) &d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	
	// allocate C in device
	Matrix d_C;
	d_C.width = C.width; d_C.height = C.height;
	size = d_C.width * d_C.height * sizeof(float);
	hipMalloc((void**) &d_C.elements, size);

	// call kernel
        dim3 dimBlock(100); // define the block size (what is the best value?) 
        dim3 dimGrid(1000); //  choose grid size depending on problem size 
		
		
	timer = NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);	
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipDeviceSynchronize();
	sdkStopTimer(&timer);
    printf("GPU Time - %f ms\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);
	
	// copy C to host
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	
	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

void MatMulCPU(Matrix A, Matrix B, Matrix C) {
	StopWatchInterface *timer;

	timer = NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);	

	for (int x = 0; x < A.width * A.height; x++) {
		float sum = 0;
		for (int i = 0; i < A.width; i++) {
			sum += A.elements[x / A.width + i] * B.elements[x % A.width + i  * A.width];
		}
		C.elements[x] = sum;
	}

	sdkStopTimer(&timer);
    printf("CPU Time - %f ms\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);
}

// matrix multiplication kernel
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = gridDim.x * blockDim.x;
    for (int x = tid; x < A.width * A.height; x += offset) {
		float sum = 0;
		for (int i = 0; i < A.width; i++) {
			sum += A.elements[(x / A.width) + i] * B.elements[(x % A.width) + (i  * A.width)];
		}
		C.elements[x] = sum;
	}
}

int main(int argc, char * const argv[])
{	
	int Height = 512;
	int Width = 1024;

	Matrix A;
	Matrix B;
	Matrix C;

	A.height = Height;
	B.height = Width;
	C.height = Height;

	A.width = Width;
	B.width = Height;
	C.width = Height;

	A.elements = new float[Width*Height];
	B.elements = new float[Width*Height];
	C.elements = new float[Width*Height];

	// fill matrices
	std::ifstream A_input;
	std::ifstream B_input;
	A_input.open("A_512x1024.txt");
	B_input.open("B_512x1024.txt");
	
	float a, b;
	A_input >> a;	
	B_input >> b;	
	int i = 0;
	while (!A_input.eof())
	{	A.elements[i] = a;
		B.elements[i] = b;
		A_input >> a;	
		B_input >> b;	
		i += 1;
	}
	A_input.close();
	B_input.close();

	MatMulGPU(A, B, C);
	//MatMulCPU(A, B, C);

	/*
	std::ofstream C_output;
	C_output.open("C.txt");
	for (int i=0; i<Width; i++)
	{	for (int j=0; j<Width; j++)
			C_output<<C.elements[i*Width+j]<<"\t";
		C_output<<endl;
	}
	*/
}
	
