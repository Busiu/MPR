#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "helper_timer.h"

__global__ void addGPU(int* a, int* b, int* c, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for (int i = tid; i < N; i += offset)
        c[i] = a[i] + b[i];
}

void addCPU(int* a, int* b, int* c, int N) {
    for (int i = 0; i < N; i++)
        c[i] = a[i] + b[i];
}

bool isVectorsEqual(int* a, int* b, int N) {
    for (int i = 0; i < N; i++) {
        if (a[i] != b[i])
            return false;
    }
    return true;
}

int main(void) {
    int *a, *b, *c, *d;
    int *dev_a, *dev_b, *dev_c;
    int N, number_of_threads_per_block, number_of_blocks;
    StopWatchInterface *timer;

    printf("Please, enter the size of an array:\n");
    scanf("%d", &N);
    printf("Please, enter the number of threads per block:\n");
    scanf("%d", &number_of_threads_per_block);
    printf("Please, enter the number of blocks:\n");
    scanf("%d", &number_of_blocks);

    a = (int*) malloc(N * sizeof(int));
    b = (int*) malloc(N * sizeof(int));
    c = (int*) malloc(N * sizeof(int));
    d = (int*) malloc(N * sizeof(int));
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = 2 * i;
    }

    // GPU ----------------------------------------------------------
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice);

    timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    addGPU <<<number_of_blocks, number_of_threads_per_block>>> (dev_a, dev_b, dev_c, N);
    hipDeviceSynchronize();
    
    sdkStopTimer(&timer);
    printf("GPU Time - %f ms\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    // --------------------------------------------------------------
    
    // CPU ----------------------------------------------------------
    timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    addCPU(a, b, d, N);

    sdkStopTimer(&timer);
    printf("CPU Time - %f ms\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);
    // --------------------------------------------------------------

    if (isVectorsEqual(c, d, N))
        printf("Vectors are the same\n");
    else
        printf("Vectors are not the same\n");

    hipFree(dev_a);
    hipFree(dev_a);
    hipFree(dev_a);
    free(a);
    free(b);
    free(c);
    free(d);

    return 0;
}